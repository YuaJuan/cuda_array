#include "hip/hip_runtime.h"
#include"cuda_need.h"

__global__ void smooth_pitch(float *data,float *out,size_t pitch,int dataSize,int winsize){	
	int x = blockIdx.y*blockDim.y + threadIdx.y;
	int y = blockIdx.x*blockDim.x + threadIdx.x;
	int temp_x, temp_y = 0;
	float sum = 0.0;
	int count = 0;
	float *row_a;
	
	if (x < dataSize&&y < dataSize){
		for (int i = 0; i < winsize; i++){
			temp_y = y + i;
			if (temp_y < dataSize)
				row_a = (float*)((char*)data + temp_y * pitch);
			else
				break;
			for (int j = 0; j < winsize; j++){
				temp_x = x+ j;
				if (temp_x < dataSize){
					sum += row_a[temp_x];
					count++;
				}
			}
			row_a = (float*)((char*)out + y*pitch);
			row_a[x] = sum;
		}
		
	}
}

void smooth2D_pre_data(char filepath[], int imgsize, float* memcpyHD_2D, float* memcpyDH_2D, float* kernel_2D, float* total_2D)
{	
	float *d_data;
	float *d_out;
	float timeDelay;

	size_t pitch;
    clock_t begintime, endtime,totalbegintime,totalendtime;

	float *data = new float[imgsize*imgsize];
	readData(filepath, data, imgsize);
	totalbegintime = clock();
//	printf("cuda_smooth2D begin.....\n");

    hipMallocPitch((void**)&d_data, &pitch, imgsize*sizeof(float), imgsize);
	hipMallocPitch((void**)&d_out, &pitch, imgsize*sizeof(float), imgsize);

	begintime = clock();
	
	hipMemcpy2D(d_data, pitch, data, imgsize*sizeof(float), imgsize*sizeof(float), imgsize, hipMemcpyHostToDevice);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyHD_2D = *memcpyHD_2D + timeDelay;
	//printf("in 2D memcpyHostToDevice time is :%.3fms\n", timeDelay);

	begintime = clock();
	// the gpu used maximum number of threads of per block:1024
	dim3 dimBlock(32, 32);
	//max of grid 2147483647
	dim3 dimGrid((imgsize + dimBlock.x - 1) / (dimBlock.x), (imgsize + dimBlock.y - 1) / (dimBlock.y));
	smooth_pitch << <dimGrid, dimBlock >> >(d_data, d_out, pitch, imgsize, WINSIZE);
	//smooth1D << <dimGrid, dimBlock >> >(d_data, d_out, DATASIZE, WINSIZE);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_2D = *kernel_2D + timeDelay;

	//printf("in 2D kernel function time :%.3fms\n", timeDelay);

	begintime = clock();
	hipMemcpy2D(data, imgsize*sizeof(float), d_out, pitch, imgsize*sizeof(float), imgsize, hipMemcpyDeviceToHost);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyDH_2D = *memcpyDH_2D + timeDelay;
	totalendtime = clock();
//	printf("in 2D memcpyDeviceToHost time is :%.3fms\n", timeDelay);
	timeDelay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_2D = *total_2D + timeDelay;
//	printf("in 2D cuda_smooth2D total time is :%.3fms\n", timeDelay);
	//printf("\n\n");
}

