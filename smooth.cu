#include "hip/hip_runtime.h"
#include"cuda_need.h"


void do_smooth(float **data,int imgsize);
void readData2D(char filePath[],float **data){
	int x = 0;
	int y = 0;
	float val = 0;
	char delimes[] = ",";
	char *split_result = NULL;
	char strLine[1024];
	FILE *fp;
	if ((fp = fopen(filePath, "r")) == NULL){
		printf("read %s file failed.\n",filePath);
		exit(-1);
	}

	while (!feof(fp)){
		fgets(strLine, 1024, fp);
			
		split_result = strtok(strLine, delimes);
		split_result = strtok(NULL, delimes);
		if (split_result == NULL)break;
		x = atoi(split_result);
	
		split_result = strtok(NULL, delimes);
		y = atoi(split_result);
	
		split_result = strtok(NULL, delimes);
		val = atof(split_result);		

		data[x][y] = val;
		//printf("%.2f", val);
	}
	fclose(fp);
}

void smooth_pre_data_cpu(char filePath[], int imgsize, float* total_cpu){
	float **data;
    clock_t start,end;
	float timeDelay;
   
	data = (float **)malloc(sizeof(float *)*imgsize);
	for (int i = 0; i < imgsize; i++)
		data[i] = (float *)malloc(sizeof(float)*imgsize);

	readData2D(filePath, data);
	start = clock();
	do_smooth(data,imgsize);
	end = clock();
	timeDelay = (double)(end - start) * 1000 / CLOCKS_PER_SEC;
	*total_cpu = *total_cpu + timeDelay;
	//printf("\nsmooth_cpu starting....\n");
	//printf("do_smooth_cpu:%.3fms\n",timeDelay);
}

void do_smooth(float **data,int imgsize){
	int sum, count;
	int x, y;
	for (int i = 0; i < imgsize; i++)
	{
		for (int j = 0; j < imgsize; j++){
			sum = 0;
			count = 0;
			for (int m = 0; m < WINSIZE; m++){
				for (int n = 0; n < WINSIZE; n++){
					x = i + m;
					y = j + n;
					if (x < imgsize&&y < imgsize){
						sum += data[x][y];
						count++;
					}
				}
			}
			data[i][j] = sum / count;
		}
	}
}
