#include"cuda_need.h"
void readData(char filePath[], float *data,int imgsize){
	int x = 0;
	int y = 0;
	float val = 0;
	char delimes[] = ",";
	char *split_result = NULL;
	char strLine[1024];
	FILE *fp;
	if ((fp = fopen(filePath, "r")) == NULL){
		printf("read %s file failed.\n",filePath);
		exit(-1);
	}

	while (!feof(fp)){
		fgets(strLine, 1024, fp);

		split_result = strtok(strLine, delimes);
		split_result = strtok(NULL, delimes);
		if (split_result == NULL)break;
		x = atoi(split_result);

		split_result = strtok(NULL, delimes);
		y = atoi(split_result);

		split_result = strtok(NULL, delimes);
		val = atof(split_result);

		data[y + x*imgsize] = val;
		//printf("%.2f", val);
	}
	fclose(fp);
}

void writeExcel(float time)
{
	FILE *fp = NULL;
	fp = fopen("D:\\data.xls", "a");
	fprintf(fp, "%.3f\t", time);
		//fprintf(fp, "%c\t%d\n", 'e', 2);
	fclose(fp);
}

void writeExcelLine(){
	FILE *fp = NULL;
	fp = fopen("D:\\data.xls", "a+");
	fprintf(fp, "\n");
	fclose(fp);
}

int putString2Csv(char str[], char filename[], int mode)
{
	FILE *_fp;
	//try to open file  
	if ((_fp = fopen(filename, "a")) == NULL)
	{
		printf("fopen called error");
		exit(-1);
	}

	int _mode = mode;

	switch (_mode)
	{
	case 1:
	{
						 fputs(str, _fp);
						 fputs("\t", _fp);
	}break;
	case 0:
	{
						   fputs("\n", _fp);
	}break;
	default:break;
	}
	if (fclose(_fp) != 0)
	{
		printf("fclose called error");
		exit(-1);
	}

	return 1;
}

void putStringToCsv(char str[],char filename[]){
	FILE *fp;
	if((fp=fopen(filename,"a"))==NULL){
		printf("fopen called error");
		exit(-1);
	}
	fputs(str,fp);
	
    fclose(fp);
}


void readData2D(char filePath[], float **data){
	int x = 0;
	int y = 0;
	float val = 0;
	char delimes[] = ",";
	char *split_result = NULL;
	char strLine[1024];
	FILE *fp;
	if ((fp = fopen(filePath, "r")) == NULL){
		printf("read %s file failed.\n", filePath);
		exit(-1);
	}

	while (!feof(fp)){
		fgets(strLine, 1024, fp);

		split_result = strtok(strLine, delimes);
		split_result = strtok(NULL, delimes);
		if (split_result == NULL)break;
		x = atoi(split_result);

		split_result = strtok(NULL, delimes);
		y = atoi(split_result);

		split_result = strtok(NULL, delimes);
		val = atof(split_result);

		data[x][y] = val;
		//printf("%.2f", val);
	}
	fclose(fp);
}