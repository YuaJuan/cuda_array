#include "hip/hip_runtime.h"
#include"cuda_need.h"

__global__ void smooth1D(float *data, float *out, int dataSize, int winsize){
	int t_x = blockIdx.y*blockDim.y + threadIdx.y;
	int t_y = blockIdx.x*blockDim.x + threadIdx.x;
	int offset = t_y*gridDim.x*blockDim.y + t_x;
	int tmp_offset = offset;
	int x = offset / dataSize;
	int y = offset%dataSize;
	float sum = 0.0;
	int count = 0;
	int tmp;

	if (x < dataSize&&y < dataSize){
		for (int i = 0; i < winsize; i++){
			tmp_offset += i*dataSize;
			for (int j = 0; j < winsize; j++){
				tmp = tmp_offset + j;
				x = tmp / dataSize;
				y = tmp%dataSize;
				if (x < dataSize&&y < dataSize){
					sum += data[tmp];
					count++;
				}
			}
		}
		out[offset] = sum / count;
	}
}


__global__ void smooth_pitch(float *data, float *out, size_t pitch, int dataSize, int winsize){
	int x = blockIdx.y*blockDim.y + threadIdx.y;
	int y = blockIdx.x*blockDim.x + threadIdx.x;
	int temp_x, temp_y = 0;
	float sum = 0.0;
	int count = 0;
	float *row_a;

	if (x < dataSize&&y < dataSize){
		for (int i = 0; i < winsize; i++){
			temp_y = y + i;
			if (temp_y < dataSize)
				row_a = (float*)((char*)data + temp_y * pitch);
			else
				break;
			for (int j = 0; j < winsize; j++){
				temp_x = x + j;
				if (temp_x < dataSize){
					sum += row_a[temp_x];
					count++;
				}
			}
			row_a = (float*)((char*)out + y*pitch);
			row_a[x] = sum;
		}

	}
}
