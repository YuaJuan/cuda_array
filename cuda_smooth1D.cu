#include "hip/hip_runtime.h"
#include"cuda_need.h"

__global__ void smooth1D(float *data, float *out, int dataSize, int winsize){
	int t_x = blockIdx.y*blockDim.y + threadIdx.y;
	int t_y = blockIdx.x*blockDim.x + threadIdx.x;
	int offset = t_y*gridDim.x*blockDim.y + t_x;
	int tmp_offset = offset;
	int x = offset/dataSize;
	int y = offset%dataSize;
	float sum = 0.0;
	int count = 0;
	int tmp;

	if (x < dataSize&&y < dataSize){
		for (int i = 0; i < winsize; i++){
			tmp_offset += i*dataSize;
			for (int j = 0; j < winsize; j++){
				tmp = tmp_offset + j;
				x = tmp / dataSize;
				y = tmp%dataSize;
				if (x < dataSize&&y < dataSize){
					sum += data[tmp];
					count++;
				}
			}	
		}
		out[offset] = sum / count;
	}
}




void smooth1D_pre_data(char filePath[], int imgsize, float* memcpyHD_1D, float* memcpyDH_1D, float* kernel_1D, float* total_1D)
{

	float *d_data;
	float *d_out;
	float timeDelay;
	clock_t begintime, endtime;
    clock_t totalbegintime,totalendtime;
	float *data = new float[imgsize*imgsize];
	readData(filePath, data, imgsize);
	totalbegintime = clock();

    //printf("\ncuda_smooth1D begin....\n");
	hipMalloc((void**)&d_data, sizeof(float)*imgsize*imgsize);
	hipMalloc((void**)&d_out, sizeof(float)*imgsize*imgsize);
	
	begintime = clock();
	hipMemcpy(d_data, data, sizeof(float)*imgsize*imgsize, hipMemcpyHostToDevice);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyHD_1D = *memcpyHD_1D + timeDelay;
//	printf("in 1D memcpyHD time is :%.3fms\n", timeDelay);

	begintime = clock();
	dim3 dimBlock(32, 32);
	dim3 dimGrid((imgsize + dimBlock.x - 1) / (dimBlock.x), (imgsize + dimBlock.y - 1) / (dimBlock.y));
	smooth1D << <dimGrid, dimBlock >> >(d_data, d_out, imgsize, WINSIZE);
	
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_1D = *kernel_1D + timeDelay;
//	printf("in 1D kernel time :%.3fms\n", timeDelay);

	begintime = clock();
	hipMemcpy(data, d_out, sizeof(float)*imgsize*imgsize, hipMemcpyDeviceToHost);
	endtime = clock();
    hipDeviceSynchronize();
	totalendtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyDH_1D = *memcpyDH_1D + timeDelay;
//	printf("1D memcpyDH time is :%.3fms\n", timeDelay);
	
	timeDelay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_1D = *total_1D + timeDelay;
   // for(int i=0;i<10;i++)
    //  printf("%.3f ",data[i]);

  //  printf("\n");
   // printf("in 1D  total time is:%.3fms\n",timeDelay);
	//printf("\n\n");

}

