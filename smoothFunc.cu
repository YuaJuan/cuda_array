#include "hip/hip_runtime.h"

#include"cuda_need.h"

void smooth1D_pre_data(char filePath[], int imgsize, float* memcpyHD_1D, float* memcpyDH_1D, float* kernel_1D, float* total_1D)
{

	float *d_data;
	float *d_out;
	float timeDelay;
	clock_t begintime, endtime;
	clock_t totalbegintime, totalendtime;
	float *data = new float[imgsize*imgsize];
	readData(filePath, data, imgsize);
	totalbegintime = clock();

	//printf("\ncuda_smooth1D begin....\n");
	hipMalloc((void**)&d_data, sizeof(float)*imgsize*imgsize);
	hipMalloc((void**)&d_out, sizeof(float)*imgsize*imgsize);

	begintime = clock();
	hipMemcpy(d_data, data, sizeof(float)*imgsize*imgsize, hipMemcpyHostToDevice);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyHD_1D = *memcpyHD_1D + timeDelay;
	//	printf("in 1D memcpyHD time is :%.3fms\n", timeDelay);

	begintime = clock();
	dim3 dimBlock(32, 32);
	dim3 dimGrid((imgsize + dimBlock.x - 1) / (dimBlock.x), (imgsize + dimBlock.y - 1) / (dimBlock.y));
	smooth1D << <dimGrid, dimBlock >> >(d_data, d_out, imgsize, WINSIZE);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_1D = *kernel_1D + timeDelay;
	//	printf("in 1D kernel time :%.3fms\n", timeDelay);

	begintime = clock();
	hipMemcpy(data, d_out, sizeof(float)*imgsize*imgsize, hipMemcpyDeviceToHost);
	endtime = clock();
	hipDeviceSynchronize();
	totalendtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyDH_1D = *memcpyDH_1D + timeDelay;
	//	printf("1D memcpyDH time is :%.3fms\n", timeDelay);

	timeDelay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_1D = *total_1D + timeDelay;
	// for(int i=0;i<10;i++)
	//  printf("%.3f ",data[i]);

	//  printf("\n");
	// printf("in 1D  total time is:%.3fms\n",timeDelay);
	//printf("\n\n");

}


void smooth2D_pre_data(char filepath[], int imgsize, float* memcpyHD_2D, float* memcpyDH_2D, float* kernel_2D, float* total_2D)
{
	float *d_data;
	float *d_out;
	float timeDelay;

	size_t pitch;
	clock_t begintime, endtime, totalbegintime, totalendtime;

	float *data = new float[imgsize*imgsize];
	readData(filepath, data, imgsize);
	totalbegintime = clock();
	//	printf("cuda_smooth2D begin.....\n");

	hipMallocPitch((void**)&d_data, &pitch, imgsize*sizeof(float), imgsize);
	hipMallocPitch((void**)&d_out, &pitch, imgsize*sizeof(float), imgsize);

	begintime = clock();

	hipMemcpy2D(d_data, pitch, data, imgsize*sizeof(float), imgsize*sizeof(float), imgsize, hipMemcpyHostToDevice);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyHD_2D = *memcpyHD_2D + timeDelay;
	//printf("in 2D memcpyHostToDevice time is :%.3fms\n", timeDelay);

	begintime = clock();
	// the gpu used maximum number of threads of per block:1024
	dim3 dimBlock(32, 32);
	//max of grid 2147483647
	dim3 dimGrid((imgsize + dimBlock.x - 1) / (dimBlock.x), (imgsize + dimBlock.y - 1) / (dimBlock.y));
	smooth_pitch << <dimGrid, dimBlock >> >(d_data, d_out, pitch, imgsize, WINSIZE);
	//smooth1D << <dimGrid, dimBlock >> >(d_data, d_out, DATASIZE, WINSIZE);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_2D = *kernel_2D + timeDelay;

	//printf("in 2D kernel function time :%.3fms\n", timeDelay);

	begintime = clock();
	hipMemcpy2D(data, imgsize*sizeof(float), d_out, pitch, imgsize*sizeof(float), imgsize, hipMemcpyDeviceToHost);
	endtime = clock();
	timeDelay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyDH_2D = *memcpyDH_2D + timeDelay;
	totalendtime = clock();
	//	printf("in 2D memcpyDeviceToHost time is :%.3fms\n", timeDelay);
	timeDelay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_2D = *total_2D + timeDelay;
	//	printf("in 2D cuda_smooth2D total time is :%.3fms\n", timeDelay);
	//printf("\n\n");
}

void smooth_pre_data_cpu(char filePath[], int imgsize, float* total_cpu){
	float **data;
	int endtime, begintime;
	clock_t start, end;
	double timedelay;
	float timeDelay;

	data = (float **)malloc(sizeof(float *)*imgsize);
	for (int i = 0; i < imgsize; i++)
		data[i] = (float *)malloc(sizeof(float)*imgsize);

	readData2D(filePath, data);
	start = clock();
	do_smooth(data, imgsize);
	end = clock();
	timeDelay = (double)(end - start) * 1000 / CLOCKS_PER_SEC;
	*total_cpu = *total_cpu + timeDelay;
	printf("\nsmooth_cpu starting....\n");
	printf("do_smooth_cpu:%.3fms\n", timeDelay);
}

void do_smooth(float **data, int imgsize){
	int sum, count;
	int x, y;
	for (int i = 0; i < imgsize; i++)
	{
		for (int j = 0; j < imgsize; j++){
			sum = 0;
			count = 0;
			for (int m = 0; m < WINSIZE; m++){
				for (int n = 0; n < WINSIZE; n++){
					x = i + m;
					y = j + n;
					if (x < imgsize&&y < imgsize){
						sum += data[x][y];
						count++;
					}
				}
			}
			data[i][j] = sum / count;
		}
	}
}


void mallocHost(char filepath[], int datasize, float *kernel_HOST, float *total_HOST){
	float *data = new float[datasize*datasize];
	float *host2dev;
	float *d_data;
	float *dev_host2dev;
	float delay;
	clock_t begintime, endtime, totalbegintime, totalendtime;

	hipSetDeviceFlags(hipDeviceMapHost);
	hipMalloc((void**)&d_data, sizeof(float)*datasize*datasize);
	readData(filepath, data, datasize);
	totalbegintime = clock();
	hipMemcpy(d_data, data, sizeof(float)*datasize*datasize, hipMemcpyHostToDevice);

	//	hipHostAlloc((void**)&data, sizeof(float)*datasize*datasize, hipHostMallocMapped | hipHostMallocWriteCombined);
	hipHostAlloc((void**)&host2dev, sizeof(float)*datasize*datasize, hipHostMallocMapped | hipHostMallocWriteCombined);
	//hipHostMallocWriteCombined�ϲ�ʽд�룬��Ч���GPU��ȡ����ڴ棬�������CPUҲ��Ҫ��ȡ����ڴ棬�ή������

	hipHostGetDevicePointer(&dev_host2dev, host2dev, 0);
	//	hipHostGetDevicePointer(&d_data, data, 0);


	begintime = clock();
	dim3 dimBlock(32, 32);
	dim3 dimGrid((datasize + dimBlock.x - 1) / (dimBlock.x), (datasize + dimBlock.y - 1) / (dimBlock.y));
	smooth1D << <dimGrid, dimBlock >> >(d_data, dev_host2dev, datasize, WINSIZE);
	hipDeviceSynchronize();
	endtime = clock();
	totalendtime = clock();
	delay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_HOST = *kernel_HOST + delay;
	printf("in function kernel_HOST:%.3f\n", delay);
	delay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_HOST = *total_HOST + delay;
	printf("in funtcion total_HOST:%.3f\n", delay);
	//hipMemcpy(data, d_data, sizeof(float)*datasize*datasize, hipMemcpyDeviceToHost);
	//	for (int i = 0; i < datasize*datasize; i++){
	//		if (i%datasize == 0)
	//			printf("\n");
	//		printf("%.3f ", host2dev[i]);
	//	}
	//	printf("\n");
	/*	for (int i = 0; i <10; i++){
	if (i%datasize == 0)
	printf("\n");
	printf("%.3f ", host2dev[i]);
	}
	printf("\n");*/
	hipFree(d_data);
	hipHostFree(dev_host2dev);

}


void mallocHostAll(char filepath[], int datasize, float *kernel_HOST, float *total_HOST){
	float *data;
	float *host2dev;
	float *d_data;
	float *dev_host2dev;
	float delay;
	clock_t begintime, endtime, totalbegintime, totalendtime;

	hipSetDeviceFlags(hipDeviceMapHost);

	hipHostAlloc((void**)&data, sizeof(float)*datasize*datasize, hipHostMallocMapped | hipHostMallocWriteCombined);
	hipHostAlloc((void**)&host2dev, sizeof(float)*datasize*datasize, hipHostMallocMapped | hipHostMallocWriteCombined);
	//hipHostMallocWriteCombined�ϲ�ʽд�룬��Ч���GPU��ȡ����ڴ棬�������CPUҲ��Ҫ��ȡ����ڴ棬�ή������

	readData(filepath, data, datasize);
	totalbegintime = clock();
	hipHostGetDevicePointer(&dev_host2dev, host2dev, 0);
	hipHostGetDevicePointer(&d_data, data, 0);


	begintime = clock();
	dim3 dimBlock(32, 32);
	dim3 dimGrid((datasize + dimBlock.x - 1) / (dimBlock.x), (datasize + dimBlock.y - 1) / (dimBlock.y));
	smooth1D << <dimGrid, dimBlock >> >(d_data, dev_host2dev, datasize, WINSIZE);
	hipDeviceSynchronize();
	endtime = clock();
	totalendtime = clock();
	delay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_HOST = *kernel_HOST + delay;
	printf("in function kernel_HOSTALL:%.3f\n", delay);
	delay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_HOST = *total_HOST + delay;
	printf("in funtcion total_HOSTALL:%.3f\n", delay);
	//hipMemcpy(data, d_data, sizeof(float)*datasize*datasize, hipMemcpyDeviceToHost);
	//	for (int i = 0; i < datasize*datasize; i++){
	//		if (i%datasize == 0)
	//			printf("\n");
	//		printf("%.3f ", host2dev[i]);
	//	}
	//	printf("\n");
	/*	for (int i = 0; i <10; i++){
	if (i%datasize == 0)
	printf("\n");
	printf("%.3f ", host2dev[i]);
	}
	printf("\n");*/
	hipFree(d_data);
	hipHostFree(dev_host2dev);

}


void mallocHostDefault(char filepath[], int datasize, float *memcpyDH_hostDefault,float *kernel_hostDefault, float *total_hostDefault){
	float *data=new float[datasize*datasize];
	float *d_data;
	float *out;
	float delay;
	clock_t begintime, endtime, totalbegintime, totalendtime;

	readData(filepath, data, datasize);
	totalbegintime = clock();
	hipMalloc((void**)&d_data, sizeof(float)*datasize*datasize);
	begintime=clock();
	hipMemcpy(d_data, data, sizeof(float)*datasize*datasize, hipMemcpyHostToDevice);
	endtime=clock();
	delay=(double)(endtime-begintime)*1000/CLOCKS_PER_SEC;

//	printf("int function mallocHostDefault memcpyHD:%.3f\n",delay);
	hipHostAlloc((void**)&out, sizeof(float)*datasize*datasize,hipHostMallocDefault);

	begintime = clock();
	dim3 dimBlock(32, 32);
	dim3 dimGrid((datasize + dimBlock.x - 1) / (dimBlock.x), (datasize + dimBlock.y - 1) / (dimBlock.y));
	smooth1D << <dimGrid, dimBlock >> >(d_data, out, datasize, WINSIZE);
	hipDeviceSynchronize();
	endtime = clock();
	delay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_hostDefault = *kernel_hostDefault + delay;
//	printf("in function mallocHostDefault kernel :%.3f\n", delay);

	begintime = clock();
	hipMemcpy(data, out, sizeof(float)*datasize*datasize, hipMemcpyDeviceToHost);
	endtime = clock();
	totalendtime = clock();
	delay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*memcpyDH_hostDefault = *memcpyDH_hostDefault + delay;
	//printf("in function mallocHostDefault memcpyDH:%.3f\n", delay);
	delay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_hostDefault = *total_hostDefault + delay;
	//printf("in function mallocHostDefault totaltime:%.3f\n", delay);

}
