#include"cuda_need.h"


int main(){
	int num = 8;
	int loop_times = 5;
	int imgsize;
	int datasize[]={500,1000,1500,2000,2500,3000,4000,5000,7000,9000,11000,13000,15000};
//
//    int datasize[]={500,11000};
	char filepath[1024];
    char csvfilepath[]="./test1.txt";	
	char strtemp[1024];
//	sprintf(strtemp, "%s\t%s\t%s\t%s\t%s\t%s\t%s\t%s\t%s\t%s\n", "imgsize", "memcpyHD_1D", "memcpyDH_1D", "kernel_1D", "total_1D",
//		"memcpyHD_2D", "memcpyDH_2D", "kernel_2D", "total_2D", "total_cpu");

   sprintf(strtemp,"%s\t%s\t%s\n","imgsize","kernel_host","total_host");
	putStringToCsv(strtemp, csvfilepath);
	
	char str_num[20];
	for (int i =8; i < 11; i++){
		float memcpyHD_1D = 0.0, memcpyDH_1D = 0.0, kernel_1D = 0.0, total_1D = 0.0;
		float memcpyHD_2D = 0.0, memcpyDH_2D = 0.0, kernel_2D = 0.0, total_2D = 0.0;
		float memcpyHD_HOST = 0.0, memcpyDH_HOST = 0.0, kernel_HOST = 0.0, total_HOST = 0.0;
		float total_cpu = 0.0;
		memset(filepath, 0, 1024);
		imgsize = datasize[i];
		strcat(filepath,"/home/ajuan/sparkArray/data/data");
		//strcat(filepath, "D:\\data\\data");
		sprintf(str_num, "%d", datasize[i]);
		strcat(filepath, str_num);
		printf("now deal with file %s\n", filepath);
		for (int i = 0; i < loop_times; i++){
		//	smooth1D_pre_data(filepath, imgsize, &memcpyHD_1D, &memcpyDH_1D, &kernel_1D, &total_1D);
		//	smooth2D_pre_data(filepath, imgsize, &memcpyHD_2D, &memcpyDH_2D, &kernel_2D, &total_2D);
		//	smooth_pre_data_cpu(filepath, imgsize,&total_cpu);	
		//	mallocHost(filepath, imgsize, &kernel_HOST, &total_HOST);
			mallocHostAll(filepath, imgsize, &kernel_HOST, &total_HOST);
			
		}

		
	//	sprintf(strtemp, "%d\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\n", imgsize, memcpyHD_1D / loop_times, memcpyDH_1D / loop_times, 
	//		kernel_1D / loop_times, total_1D / loop_times,memcpyHD_2D / loop_times, memcpyDH_2D / loop_times, 
	//		kernel_2D / loop_times, total_2D / loop_times, total_cpu / loop_times);
		sprintf(strtemp,"%d\t%.3f\t%.3f\n",imgsize,kernel_HOST/loop_times,total_HOST/loop_times);
		putStringToCsv(strtemp, csvfilepath);


/*		printf("#####################################################\n");
		printf("memcpyHD_1D time is :%.3f\n", memcpyHD_1D / loop_times);
		printf("kernel1D time is :%.3f\n", kernel_1D / loop_times);
		printf("memcpyDH_1D time is :%.3f\n", memcpyDH_1D / loop_times);
		printf("total_1D time is :%.3f\n\n", total_1D / loop_times);
		

		printf("#####################################################\n");
		printf("memcpyHD_2D time is :%.3f\n", memcpyHD_2D / loop_times);
		printf("kernel2D time is :%.3f\n", kernel_2D / loop_times);
		printf("memcpyDH_2D time is :%.3f\n", memcpyDH_2D / loop_times);
		printf("total_2D time is :%.3f\n\n", total_2D / loop_times);

		printf("#####################################################\n");
		printf("total time is :%.3f\n", total_cpu / loop_times);
*/
		printf("\n*********************************************\n");
		printf("kernel_host:%.3f\n", kernel_HOST/loop_times);
		printf("total_host:%.3f\n", total_HOST/loop_times);
	}
	
}





