#include "hip/hip_runtime.h"
#include"cuda_need.h"


void mallocHostAll(char filepath[], int datasize, float *kernel_HOST, float *total_HOST){
	float *data;
	float *host2dev;
	float *d_data;
	float *dev_host2dev;
	float delay;
	clock_t begintime, endtime, totalbegintime, totalendtime;

	hipSetDeviceFlags(hipDeviceMapHost);

	hipHostAlloc((void**)&data, sizeof(float)*datasize*datasize, hipHostMallocMapped | hipHostMallocWriteCombined);
	hipHostAlloc((void**)&host2dev, sizeof(float)*datasize*datasize, hipHostMallocMapped | hipHostMallocWriteCombined);
	//hipHostMallocWriteCombined�ϲ�ʽд�룬��Ч���GPU��ȡ����ڴ棬�������CPUҲ��Ҫ��ȡ����ڴ棬�ή������

	readData(filepath, data, datasize);
	totalbegintime = clock();
	hipHostGetDevicePointer(&dev_host2dev, host2dev, 0);
	hipHostGetDevicePointer(&d_data, data, 0);

	
	begintime = clock();
	dim3 dimBlock(32, 32);
	dim3 dimGrid((datasize + dimBlock.x - 1) / (dimBlock.x), (datasize + dimBlock.y - 1) / (dimBlock.y));
	smooth1D << <dimGrid, dimBlock >> >(d_data, dev_host2dev, datasize, WINSIZE);
	hipDeviceSynchronize();
	endtime = clock();
	totalendtime = clock();
	delay = (double)(endtime - begintime) * 1000 / CLOCKS_PER_SEC;
	*kernel_HOST = *kernel_HOST + delay;
	printf("in function kernel_HOSTALL:%.3f\n", delay);
	delay = (double)(totalendtime - totalbegintime) * 1000 / CLOCKS_PER_SEC;
	*total_HOST = *total_HOST + delay;
	printf("in funtcion total_HOSTALL:%.3f\n", delay);
	//hipMemcpy(data, d_data, sizeof(float)*datasize*datasize, hipMemcpyDeviceToHost);
//	for (int i = 0; i < datasize*datasize; i++){
//		if (i%datasize == 0)
//			printf("\n");
//		printf("%.3f ", host2dev[i]);
//	}
//	printf("\n");
/*	for (int i = 0; i <10; i++){
		if (i%datasize == 0)
			printf("\n");
		printf("%.3f ", host2dev[i]);
	}
	printf("\n");*/
	hipFree(d_data);
	hipHostFree(dev_host2dev);

}
